#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <strings.h>
#include "timerc.h"

/**
Prof. Bento
04/02/19
Programmer: Edsel Rudy
Course: CSCI 3396

Description: This program aims to parallelize the Range Minimum Query algorithm

**/

__global__ void warmup(){

}

//function to get right child
int get_right(int index, int NUM_NODES){
	
	if(index != -1 && ((2*index) + 1)<= NUM_NODES){
		return (2*index) + 1;
	}

	return -1;
}

	//function to get left child
int get_left(int index, int NUM_NODES){
	
	if(index != -1 && ((2*index)<= NUM_NODES)){
		return (2*index);
	}

	return -1;
}

//parallelize the merge list
void merge_list(int* dest_list, int dest_list_length, int* array1, int array1_length, int* array2){

	/*
	//for debugging
	printf("Copying: ");
	for(int i = 1; i <= array1_length;i++){
		printf("%d,", array1[i]);
	}

	printf(" and ");

	for(int i = 1; i <= array2_length;i++){
		printf("%d,", array2[i]);
	}

	printf(": \n");
	*/
	
	int x = 1;
	for(int i = 1; i <= array1_length;i++){
		dest_list[x] = array1[i];
		x++; 
	}

	x = 1;
	for(int i = array1_length+1; i <= dest_list_length;i++){

		int last_elem = array1[array1_length];

		if(last_elem <= array2[x]){
			dest_list[i] = last_elem;
		}else{
			dest_list[i] = array2[x];
		}

		x++;
	}

	/*
	//for debugging
	for(int i = 1; i <= dest_list_length;i++){
		printf("%d,", dest_list[i]);
	}
	printf(": \n");
	*/
}

//returns the tree array offset for a specific level and node
int chooseNode(int numNodes,int level, int node){

	double height = ceil(log2(numNodes));

	int offset = (numNodes/pow(2,height-level)) + node;

	return offset;
}

//num of children for each node
int numOfChildren(int node, int numInputs){

	int level = floor(log2(node));
	int numChildren = numInputs/ pow(2,level);

	//printf("Node: %d Num Children: %d\n", node, numChildren);

	return numChildren;
}

void print_list(int** list, int numInputs){

	int numNodes = 2 * numInputs - 1;	


	for(int i = 1; i <= numNodes; i++){
		
		printf("Index %d: ", i);

		for(int x = 1; x <= (numOfChildren(i,numInputs)); x++){

			printf("%d, ", list[i][x]);
		}

		printf("\n");
	}
}

void fillInLists(int* inputArray, int numNodes, int numInputs, int** prefixTree, int** suffixTree){

	double height = ceil(log2(numNodes));

	//printf("\nTesting fill leaves\n");


	for(int i = 1; i <= pow(2,height -1); i++){
		int index = chooseNode(numNodes,height-1,i);

		int * prefixList = prefixTree[index];
		prefixList[1] = inputArray[i-1];

		int* suffixList = suffixTree[index];
		suffixList[1] = inputArray[i-1];
	}

	for(int i = height-2; i>=0; i--){
		for(int x = 1; x <= pow(2,i);x++){

			int index = chooseNode(numNodes,i,x);
			//int *prefixList = prefixTree[index];
			//int *suffixList = suffixTree[index];

			int left_index = get_left(index, numNodes);
			int right_index = get_right(index, numNodes);

			float merge_list_time = 0;
			
			//do this so that the timer only counts once
			if((i == 0)){
				cstart();
			}

			merge_list(prefixTree[index], numOfChildren(index,numInputs), prefixTree[left_index], 
				numOfChildren(left_index,numInputs),prefixTree[right_index]);

			if((i == 0)){
				cend(&merge_list_time);
				printf("Merge List Time: %f\n",merge_list_time);
			}

			merge_list(suffixTree[index], numOfChildren(index,numInputs), suffixTree[right_index],
			 numOfChildren(right_index,numInputs), suffixTree[left_index]);
		}
	}
}

int LevelOrder_LCA(int num1, int num2){

	int binaryLength = log2(num1);
	int pos = binaryLength;
	int mask = 1 << binaryLength;

	//printf("\nMask: %d, Binary Length: %d\n", mask, binaryLength);
	
	int num1Offset = mask & num1;
	int num2Offset = mask & num2;
	
	while((num2Offset ^ num1Offset) == 0){
		// printf("Index %d: same, Mask: %d,Num 1 Off: %d, Num 2 Off: %d\n"
		// 	,pos, mask, num1Offset, num2Offset);

		pos--;
		mask = mask >> 1;

		num1Offset = mask & num1;
		num2Offset = mask & num2;
	}
	pos++;



	//printf("Pos of change: %d\n Calculating LCA: \n", pos);
	int result = 0;
	int power = 0;
	mask = 1 << pos;

	for(int i = pos; i <= binaryLength;i++){
		
		if((mask & num1) != 0){
			//printf("incrementing result\n");
			result += pow(2,power);
		}

		// printf("Index: %d, Mask: %d,Power: %d, Pos and Num: %d,Result: %d\n"
		// 	,i, mask, power,(mask & num1),result);

		mask = mask << 1;
		power++;
	}
	
	return result;
}

//do this one
int shiftBits(int num, int offset){

	if(num == 0){
		return 1;
	}

	int binaryLength = log2(num);
	int length = binaryLength - offset;

	int mask = 1;
	int result = 0;

	//printf("Length: %d, Offset: %d", binaryLength,length);

	for(int i = 0; i <= length;i++){

		if((mask & num) != 0){
			result += pow(2,i);
		}

		mask = mask << 1;
	}

	//printf("\nAfter Bit Shift: %d by %d bits\n", result,offset);

	result++;

	return result;
}

int RMQ(int index1, int index2, int numInputs,int** prefixTree, int** suffixTree){

	int minQuery = 0;
	int numNodes = 2* numInputs - 1;
	double height = ceil(log2(numNodes));
	

	//if index1 and 2 is right next to each other, only compare the to numbers
	if(abs(index2 - index1) == 1){
		
		int num1 = prefixTree[chooseNode(numNodes,height-1,index1 + 1)][1];
		int num2 = prefixTree[chooseNode(numNodes,height-1,index2 + 1)][1]; 
		
		if(num1 <= num2){

			return num1;
		}else{

			return num2;
		}
	}

	if(index1 == index2){
		return prefixTree[chooseNode(numNodes,height-1,index1 + 1)][1];
	}

	
	int leftIndex = chooseNode(numNodes, height-1, index1 + 1);
	int rightIndex = chooseNode(numNodes, height-1, index2 + 1);

	int LCA_index = LevelOrder_LCA(leftIndex,rightIndex);
	int leftChild = get_left(LCA_index,numNodes);
	int rightChild = get_right(LCA_index,numNodes);


	//printf("leftIndex: %d, rightIndex: %d, LCA_index: %d\n",leftIndex,rightIndex, LCA_index);
	
	/**
	printf("Prefix List: \n");
	print_list(prefixTree, numInputs);

	printf("\nSuffix List: \n");
	print_list(suffixTree, numInputs);
	**/

	//change to shift bits
	int level_left = floor(log2(leftChild));
	int index1_suffix = shiftBits(index1,level_left);

	//printf("Left Child Index: %d ",index1_suffix);

	int level_right = floor(log2(rightChild));
	int index2_prefix = shiftBits(index2,level_right);

	//printf("Right Child Index: %d\n", index2_prefix);

	int minSuffix = suffixTree[leftChild][index1_suffix];
	int minPrefix = prefixTree[rightChild][index2_prefix];

	printf("Min Suffix: %d from Node %d, Min Prefix: %d from Node %d\n"
		,minSuffix,leftChild,minPrefix,rightChild);

	if(minPrefix <= minSuffix){
		minQuery = minPrefix;
	}
	else{
		minQuery = minSuffix;
	}

	return minQuery;
}

int main(){

	/*
		Range Minima Query:
									 h:
					   1              - 0
				  /        \
				 2          3         - 1
			   /  \       /   \
		      4    5     6     7	  - 2
			 / \  / \   / \   / \
			8  9 10 11 12 13 14 15    - 3

		input array:
		5,10,3,4,7,1,8,2

		All arrays will be indexed at 1
	*/

	/**
	//Test RMQ query correctness
	int input[] = {5,10,4,3,7,1,8,2};
	
	int numInputs = sizeof(input)/sizeof(int);
	
	int numNodes = 2 * numInputs - 1;	
	
	
	printf("\nInput Array: ");
	for(int i = 0; i < numInputs;i++){
		printf("%d, ",input[i] );
	}
	printf("\n");
	**/

	//test log of input sizes for report
	int numInputs = 1<<14;
	int input[numInputs];

	for(int i = 0; i < numInputs; i++){
		input[i] = i;
		
		if(i == numInputs/2){
			input[i] = 1;
		}
	}
	int numNodes = 2 * numInputs - 1;

	warmup<<<1,1>>>();

	float build_list_time;
	cstart();

	//RMQ Preprocessing
	int** prefixTree = (int**)malloc((numNodes+1) * sizeof(int*));

	for(int i = 1; i <= numNodes;i++){
		prefixTree[i] = (int*) malloc((numOfChildren(i,numInputs)+1) * sizeof(int*));
	}

	int** suffixTree = (int**)malloc((numNodes+1) * sizeof(int*));

	for(int i = 1; i <= numNodes;i++){
		suffixTree[i] = (int*) malloc((numOfChildren(i,numInputs)+1) * sizeof(int*));
	}

	cend(&build_list_time);
	printf("Build List Time: %f\n",build_list_time);

	float fill_list_time;
	cstart();
	fillInLists(input,numNodes, numInputs, prefixTree, suffixTree);
	cend(&fill_list_time);
	
	printf("Fill List Time: %f\n",fill_list_time);


	//test RMQ query 
	int index1 = numInputs/4;
	int index2 = numInputs-2;
	printf("\nMin from index: %d to %d\n", index1,index2);

	float RMQ_query_time;

	cstart();
	int RangeMinima = RMQ(index1,index2,numInputs,prefixTree,suffixTree);
	cend(&RMQ_query_time);

	printf("Min Query: %d\n" ,RangeMinima);

	printf("RMQ Query Time: %f\n",RMQ_query_time);
	/**
	//test RMQ query correctness
	index1 = 0;
	index2 = 2;

	printf("\nMin from %d to %d\n", index1,index2);
	RangeMinima = RMQ(index1,index2,numInputs,prefixTree,suffixTree);
	printf("Min Query: %d\n" ,RangeMinima);

	index1 = 0;
	index2 = 4;

	printf("\nMin from %d to %d\n", index1,index2);
	RangeMinima = RMQ(index1,index2,numInputs,prefixTree,suffixTree);
	printf("Min Query: %d\n" ,RangeMinima);

	index1 = 4;
	index2 = 6;

	printf("\nMin from %d to %d\n", index1,index2);
	RangeMinima = RMQ(index1,index2,numInputs,prefixTree,suffixTree);
	printf("Min Query: %d\n" ,RangeMinima);
	**/
	
	return 0;
}